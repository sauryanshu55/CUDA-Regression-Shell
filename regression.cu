#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include<unistd.h>
#include <math.h>
#include <gsl/gsl_cdf.h>
#include "kernels.cuh"
#include "regression_info.cuh"

#define MAX_VARIABLES 3
#define MAX_DATA_POINTS 1000
#define MAX_VARIABLE_NAME_LENGTH 50
#define MAX_COMMAND_LENGTH 100

data_t Data;
bool data_primed = false;

calculationInfo_t globalCalculationInfo;
betaCoefficients_t betaCoefficients;
standardErrors_t standardErrors;
pValues_t pValues;
varIndex_t indexes;

// Function to print the imported data
void printCSVData(int* data) {
    printf("Variable Names:\n");
    for (int i = 0; i < Data.numVars; i++) {
        printf("%s\t", Data.variableNames[i]);
    }

    printf("\n");

    for (int j = 0; j < Data.numObservations; j++) {
        for (int i = 0; i < Data.numVars; i++) {
            printf("%d\t", data[j * Data.numVars + i]);
        }
        printf("\n");
    }
}

// Function to read CSV file and initialize data array
int readCSV(const char *filename) {
    FILE *file = fopen(filename, "r");

    if (file == NULL) {
        return -1;
    }

    // Read the first row to get variable names
    char line[1024];
    if (fgets(line, sizeof(line), file) != NULL) {
        // Tokenize the line to get variable names
        char *token = strtok(line, ",");
        int variableIndex = 0;

        while (token != NULL && variableIndex < MAX_VARIABLES) {
            // Remove leading and trailing whitespaces
            sscanf(token, " %[^ \t\n]", Data.variableNames[variableIndex]);

            // Print variable names if needed
            // printf("Variable %d: %s\n", variableIndex + 1, Data.variableNames[variableIndex]);

            token = strtok(NULL, ",");
            variableIndex++;
        }
        Data.numVars = variableIndex;
    }

    // Read the rest of the file to get numerical data
    int dataPointIndex = 0;
    while (fgets(line, sizeof(line), file) != NULL && dataPointIndex < MAX_DATA_POINTS) {
        char *token = strtok(line, ",");
        int variableIndex = 0;

        while (token != NULL && variableIndex < MAX_VARIABLES) {
            sscanf(token, " %d", &Data.data[dataPointIndex * Data.numVars + variableIndex]);

            // Print data if needed
            // printf("%s: %d\n", Data.variableNames[variableIndex], data[dataPointIndex * Data.numVars + variableIndex]);

            token = strtok(NULL, ",");
            variableIndex++;
        }

        dataPointIndex++;
    }
    Data.numObservations = dataPointIndex;

    fclose(file);
    return 0;
}


int copyDataToGPU(){
    int *gpu_data, *gpu_data_cpy;
    
    hipMalloc((void**)&gpu_data,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int));
    hipMalloc((void**)&gpu_data_cpy,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int));

    hipMemcpy(gpu_data,Data.data,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int),hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = ((MAX_VARIABLES*MAX_DATA_POINTS)+ blockSize - 1) / blockSize;

    copyArrayKernel<<<gridSize,blockSize>>>(gpu_data,gpu_data_cpy,(MAX_VARIABLES*MAX_DATA_POINTS));
    
    hipDeviceSynchronize();

    hipMemcpy(Data.data_cpy,gpu_data_cpy,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int),hipMemcpyDeviceToHost);
    
    // printCSVData(data_cpy);
    hipFree(gpu_data);
    hipFree(gpu_data_cpy);
    return 1;
}

int createdIndexedData(){
    int *gpu_data, *gpu_data_cpy;
    
    hipMalloc((void**)&gpu_data,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int));
    hipMalloc((void**)&gpu_data_cpy,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int));

    hipMemcpy(gpu_data,Data.data,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int),hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = ((MAX_VARIABLES*MAX_DATA_POINTS)+ blockSize - 1) / blockSize;

    createdIndexedDataKernel<<<gridSize,blockSize>>>(gpu_data,gpu_data_cpy,0,indexes.zerothIndex,(MAX_VARIABLES*MAX_DATA_POINTS));
    createdIndexedDataKernel<<<gridSize,blockSize>>>(gpu_data,gpu_data_cpy,1,indexes.firstIndex,(MAX_VARIABLES*MAX_DATA_POINTS));
    createdIndexedDataKernel<<<gridSize,blockSize>>>(gpu_data,gpu_data_cpy,2,indexes.secondIndex,(MAX_VARIABLES*MAX_DATA_POINTS));
    
    hipDeviceSynchronize();

    hipMemcpy(Data.data_cpy,gpu_data_cpy,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int),hipMemcpyDeviceToHost);

    hipFree(gpu_data);
    hipFree(gpu_data_cpy);
    return 1;
}

int calculateVarSquared(int xvar){
    int *gpu_data;
    int *gpu_result;
    int sum=0;

    hipMalloc((void**)&gpu_data,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int));
    hipMemcpy(gpu_data,Data.data_cpy,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int),hipMemcpyHostToDevice);

    hipMalloc((void**)&gpu_result,sizeof(int));
    hipMemcpy(gpu_result,&sum,sizeof(int),hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = ((MAX_VARIABLES*MAX_DATA_POINTS)+ blockSize - 1) / blockSize;

    calculateSquareKernel<<<gridSize,blockSize>>>(gpu_data,gpu_result,(MAX_VARIABLES*MAX_DATA_POINTS),xvar);

    hipDeviceSynchronize();
    hipMemcpy(&sum,gpu_result,sizeof(int),hipMemcpyDeviceToHost);

    hipFree(gpu_data);
    hipFree(gpu_result);

    if(xvar==1)  globalCalculationInfo.sumSquaredX1=sum;
    if(xvar==2) globalCalculationInfo.sumSquaredX2=sum;

    return 0;
}


int calculateSumOfProductSquared(int var1, int var2){
    int *gpuData,*gpuResult;
    int sum=0;

    hipMalloc((void**)&gpuData,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int));
    hipMemcpy(gpuData,Data.data_cpy,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int),hipMemcpyHostToDevice);

    hipMalloc((void**)&gpuResult,sizeof(int));
    hipMemcpy(gpuResult,&sum,sizeof(int),hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = ((MAX_VARIABLES*MAX_DATA_POINTS)+ blockSize - 1) / blockSize;

    calculateSumOfProductSquaredKernel<<<gridSize,blockSize>>>(gpuData,gpuResult,(MAX_VARIABLES*MAX_DATA_POINTS),var1,var2);
    
    hipDeviceSynchronize();

    hipMemcpy(&sum,gpuResult,sizeof(int),hipMemcpyDeviceToHost);

    hipFree(gpuResult);
    hipFree(gpuData);

    if ((var1==0)&&(var2==1)) globalCalculationInfo.sumX1Y=sum;
    if ((var1==0)&&(var2==2)) globalCalculationInfo.sumX2Y=sum;
    if ((var1==1)&&(var2==2)) globalCalculationInfo.sumX1X2=sum;

    return 0;
}

int calculateVarSum(int var){
    int *gpu_data;
    int *gpu_result;
    int sum=0;

    hipMalloc((void**)&gpu_data,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int));
    hipMemcpy(gpu_data,Data.data_cpy,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int),hipMemcpyHostToDevice);

    hipMalloc((void**)&gpu_result,sizeof(int));
    hipMemcpy(gpu_result,&sum,sizeof(int),hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = ((MAX_VARIABLES*MAX_DATA_POINTS)+ blockSize - 1) / blockSize;

    calculateVarSumKernel<<<gridSize,blockSize>>>(gpu_data,gpu_result,(MAX_VARIABLES*MAX_DATA_POINTS),var);

    hipDeviceSynchronize();
    hipMemcpy(&sum,gpu_result,sizeof(int),hipMemcpyDeviceToHost);

    hipFree(gpu_data);
    hipFree(gpu_result);

    if (var==0) globalCalculationInfo.sumY=sum;
    if (var==1) globalCalculationInfo.sumX1=sum;
    if (var==2) globalCalculationInfo.sumX2=sum;
    return 0;
}

int predictModel(){
    int *gpuData;
    int *gpuPredictions;
    int *gpuResiduals;

    hipMalloc((void**)&gpuData,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int));
    hipMalloc((void**)&gpuPredictions,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int));
    hipMalloc((void**)&gpuResiduals,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int));

    hipMemcpy(gpuData,Data.data_cpy,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int),hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = ((MAX_VARIABLES*MAX_DATA_POINTS)+ blockSize - 1) / blockSize;

    predictModelKernel<<<gridSize,blockSize>>>(gpuData,gpuPredictions,gpuResiduals,betaCoefficients.beta_0,betaCoefficients.beta_1,betaCoefficients.beta_2,MAX_VARIABLES,Data.numObservations);

    hipDeviceSynchronize();

    hipMemcpy(Data.predictions,gpuPredictions,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(globalCalculationInfo.residuals,gpuResiduals,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int),hipMemcpyDeviceToHost);
    

    hipFree(gpuData);
    hipFree(gpuPredictions);
    hipFree(gpuResiduals);
    return 0;
}

int calculateStandardErrors(){
    int *gpuResiduals, *gpuResidualSum;
    int *gpuVarianceArr;

    double *gpuVarianceResiduals;
    double varianceResiduals =0;

    int sum=0;

    hipMalloc((void**)&gpuVarianceResiduals,sizeof(double));
    hipMemcpy(gpuVarianceResiduals,&varianceResiduals,sizeof(double),hipMemcpyHostToDevice);

    hipMalloc((void**)&gpuResiduals,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int));
    hipMemcpy(gpuResiduals,globalCalculationInfo.residuals,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int),hipMemcpyHostToDevice);

    hipMalloc((void**)&gpuResidualSum,sizeof(int));
    hipMemcpy(gpuResidualSum,&sum,sizeof(int),hipMemcpyHostToDevice);

    hipMalloc((void**)&gpuVarianceArr,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int));

    int blockSize = 256;
    int gridSize = ((MAX_VARIABLES*MAX_DATA_POINTS)+ blockSize - 1) / blockSize;

    calculateStandardErrorsKernel<<<gridSize,blockSize>>>(gpuResiduals,gpuResidualSum,gpuVarianceArr,gpuVarianceResiduals,Data.numObservations,(MAX_VARIABLES*MAX_DATA_POINTS));

    hipDeviceSynchronize();
    hipMemcpy(&sum,gpuResidualSum,sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(&varianceResiduals,gpuVarianceResiduals,sizeof(double),hipMemcpyDeviceToHost);

    globalCalculationInfo.residualVariance=varianceResiduals;

    hipFree(gpuVarianceResiduals);
    hipFree(gpuResidualSum);
    hipFree(gpuResiduals);

    return 0;
}

int calculateVarVariance(int var){
    int *gpuData, *gpuNumeratorSumArr;
    double *gpuVarVariance;

    double variance=0;

    hipMalloc((void**)&gpuData,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int));
    hipMemcpy(gpuData,Data.data_cpy,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int),hipMemcpyHostToDevice);

    hipMalloc((void**)&gpuVarVariance,sizeof(double));
    hipMemcpy(gpuVarVariance,&variance,sizeof(double),hipMemcpyHostToDevice);

    hipMalloc((void**)&gpuNumeratorSumArr,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int));

    int blockSize = 256;
    int gridSize = ((MAX_VARIABLES*MAX_DATA_POINTS)+ blockSize - 1) / blockSize;

    int varMean = 0;
    if (var==0) varMean=globalCalculationInfo.sumY/Data.numObservations;
    if (var==1) varMean=globalCalculationInfo.sumX1/Data.numObservations;
    if (var==2) varMean=globalCalculationInfo.sumX2/Data.numObservations;

    calculateVarVarianceKernel<<<gridSize,blockSize>>>(gpuData,gpuNumeratorSumArr,gpuVarVariance,
                                                        Data.numObservations,var,varMean,
                                                        (MAX_VARIABLES*MAX_DATA_POINTS));

    hipDeviceSynchronize();
    hipMemcpy(&variance,gpuVarVariance,sizeof(double),hipMemcpyDeviceToHost);


    if (var==0) globalCalculationInfo.varianceY=variance;
    if (var==1) globalCalculationInfo.varianceX1=variance;
    if (var==2) globalCalculationInfo.varianceX2=variance;

    hipFree(gpuData);
    hipFree(&gpuNumeratorSumArr);
    hipFree(&gpuVarVariance);

    return 0;
}

int calculateBetas(){
    calculationInfo_t cInfo=globalCalculationInfo;

    double x1=cInfo.sumSquaredX1-(pow(cInfo.sumX1,2)/Data.numObservations);
    double x2=cInfo.sumSquaredX2-(pow(cInfo.sumX2,2)/Data.numObservations);
    double x1y=cInfo.sumX1Y-(cInfo.sumX1*cInfo.sumY)/Data.numObservations;
    double x2y=cInfo.sumX2Y-(cInfo.sumX2*cInfo.sumY)/Data.numObservations;
    double x1x2=cInfo.sumX1X2-(cInfo.sumX1*cInfo.sumX2)/Data.numObservations;

    betaCoefficients.beta_1=((x2*x1y)-(x1x2*x2y))/((x1*x2)-pow(x1x2,2));
    betaCoefficients.beta_2=((x1*x2y)-(x1x2*x1y))/((x1*x2)-pow(x1x2,2));

    betaCoefficients.beta_0=(cInfo.sumY/Data.numObservations)-
                                (betaCoefficients.beta_1*(cInfo.sumX1/Data.numObservations))-
                                (betaCoefficients.beta_2*(cInfo.sumX2/Data.numObservations));

    return 0;  
}


void calculatePValues(){
    double t_Stat_beta_0=betaCoefficients.beta_0/standardErrors.beta_0_stderr;
    double t_Stat_beta_1=betaCoefficients.beta_1/standardErrors.beta_1_stderr;
    double t_Stat_beta_2=betaCoefficients.beta_0/standardErrors.beta_2_stderr;

    int degreesOfFreedom=Data.numObservations-3;

    pValues.beta_0_pVal=2*(1-gsl_cdf_tdist_P(fabs(t_Stat_beta_0),degreesOfFreedom));
    pValues.beta_1_pVal=2*(1-gsl_cdf_tdist_P(fabs(t_Stat_beta_1),degreesOfFreedom));
    pValues.beta_2_pVal=2*(1-gsl_cdf_tdist_P(fabs(t_Stat_beta_2),degreesOfFreedom));
    
}


int printRegressionResults(){
    size_t boundarySize=70;
    char boundary[boundarySize];
    memset(boundary,'-',boundarySize-1);
    boundary[boundarySize-1]='\0';

    printf("\nOutput: %s\n%s\n",Data.variableNames[indexes.zerothIndex],boundary);
    printf("Var         Coeff         Variance         StdErr         P(|T|>t)\n %s\n",boundary);
    printf("cons    %lf    %lf    %lf    %lf\n",betaCoefficients.beta_0,globalCalculationInfo.varianceY,standardErrors.beta_0_stderr,pValues.beta_0_pVal);
    printf("%s      %lf    %lf    %lf    %lf\n",Data.variableNames[indexes.firstIndex],betaCoefficients.beta_1,globalCalculationInfo.varianceX1,standardErrors.beta_1_stderr,pValues.beta_1_pVal);
    printf("%s      %lf    %lf    %lf    %lf\n\n",Data.variableNames[indexes.secondIndex],betaCoefficients.beta_2,globalCalculationInfo.varianceX2,standardErrors.beta_2_stderr,pValues.beta_2_pVal);
    return 0;
}

int runRegression(){
    printf("Running Regression...\n");
    calculateVarSquared(1);
    calculateVarSquared(2);

    calculateSumOfProductSquared(0,1);
    calculateSumOfProductSquared(0,2);
    calculateSumOfProductSquared(1,2);

    calculateVarSum(0);
    calculateVarSum(1);
    calculateVarSum(2);

    calculateBetas();

    predictModel();

    calculateStandardErrors();
    
    calculateVarVariance(0);
    calculateVarVariance(1);
    calculateVarVariance(2);

    standardErrors.beta_0_stderr=sqrt(globalCalculationInfo.residualVariance*globalCalculationInfo.varianceY);
    standardErrors.beta_1_stderr=sqrt(globalCalculationInfo.residualVariance*globalCalculationInfo.varianceX1);
    standardErrors.beta_2_stderr=sqrt(globalCalculationInfo.residualVariance*globalCalculationInfo.varianceX2);

    calculatePValues();

    printRegressionResults();
    return 1;
}

void assignVariableIndexes(char* givenY, char*  givenX1, char* givenX2){

    if (strcmp(Data.variableNames[0],givenY)==0) indexes.zerothIndex=0;
    if (strcmp(Data.variableNames[1],givenY)==0) indexes.zerothIndex=1;
    if (strcmp(Data.variableNames[2],givenY)==0) indexes.zerothIndex=2;

    if (strcmp(Data.variableNames[0],givenX1)==0) indexes.firstIndex=0;
    if (strcmp(Data.variableNames[1],givenX1)==0) indexes.firstIndex=1;
    if (strcmp(Data.variableNames[2],givenX1)==0) indexes.firstIndex=2;

    if (strcmp(Data.variableNames[0],givenX2)==0) indexes.secondIndex=0;
    if (strcmp(Data.variableNames[1],givenX2)==0) indexes.secondIndex=1;
    if (strcmp(Data.variableNames[2],givenX2)==0) indexes.secondIndex=2;
}

int executeCommand(char command[]) {
    // exit
    if (strcmp(command, "e") == 0) {
        printf("Exiting program\n");
        return -1;
    }

    // Check if the first part of the command is "load"
    if (strncmp(command, "load", 4) == 0) {
        char file_loc[100];
        // Extract the rest of the command string after "load"
        sscanf(command, "%*s %s", file_loc);

        // successful read=0
        if (readCSV(file_loc) == 0) {
            printf("Read CSV file from: %s \n", file_loc);
            data_primed = true;
        } else {
            printf("No such CSV file exists AND/OR Error in reading CSV File\n");
        }
        return 1;
    }

    // view
    if (strcmp(command, "view") == 0) {
        if (data_primed) {
            printCSVData(Data.data);
            printf("Number of vars: %d\nNumber of observations: %d\n",Data.numVars,Data.numObservations);
        } else {
            printf("Data is not loaded yet\n");
        }
        return 1;
    }


    if (strncmp(command, "reg ", 4) == 0) {
        if (data_primed){
            char givenY[MAX_VARIABLE_NAME_LENGTH], givenX1[MAX_VARIABLE_NAME_LENGTH], givenX2[MAX_VARIABLE_NAME_LENGTH];  
        // Extract the next three variables
            int offset = 4;  // Skip the "reg " part
            int count = sscanf(command + offset, "%49s %49s %49s", givenY, givenX1, givenX2);

            if (count == 3) {
                assignVariableIndexes(givenY, givenX1, givenX2);
                createdIndexedData();
                runRegression();
                return 1;
            } else {
                // Invalid format
                printf("Invalid command format.\n");
                return 1;
            }
            } else {
            // Conditions not met
                printf("Invalid command format.\n");
                return 1;
            }

    }
    // Unrecognized command
    return 0;
}

int main() {
    char command[MAX_COMMAND_LENGTH];

    while (1) {
        // Print a prompt
        printf("$$> ");

        // Read a command from the user
        if (fgets(command, MAX_COMMAND_LENGTH, stdin) == NULL) {
            perror("Error reading command");
            exit(EXIT_FAILURE);
        }

        // Remove the newline character from the end of the command
        size_t length = strlen(command);
        if (length > 0 && command[length - 1] == '\n') {
            command[length - 1] = '\0';
        }

        // Execute the command using the system function
        int result = executeCommand(command);

        // Check if the command execution was successful
        if (result == 0) {
            printf("Unrecognized command\n");
        }

        // exit
        if (result == -1) {
            return 0;
        }
    }
    return 0;
}

