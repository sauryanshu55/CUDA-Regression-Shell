#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "kernels.cuh"
#include "regression_info.cuh"

#define MAX_VARIABLES 3
#define MAX_DATA_POINTS 1000
#define MAX_VARIABLE_NAME_LENGTH 50
#define MAX_COMMAND_LENGTH 100

data_t Data;
bool data_primed = false;

calculationInfo_t globalCalculationInfo;
regressionInfo_t globalRegressionInfo;

// Function to print the imported data
void printCSVData(int* data) {
    printf("Variable Names:\n");
    for (int i = 0; i < Data.numVars; i++) {
        printf("%s\t", Data.variableNames[i]);
    }

    printf("\n");

    for (int j = 0; j < Data.numObservations; j++) {
        for (int i = 0; i < Data.numVars; i++) {
            printf("%d\t", data[j * Data.numVars + i]);
        }
        printf("\n");
    }
}

// Function to print the imported data
void printCSVData_d(double* data) {
    printf("Variable Names:\n");
    for (int i = 0; i < Data.numVars; i++) {
        printf("%s\t", Data.variableNames[i]);
    }

    printf("\n");

    for (int j = 0; j < Data.numObservations; j++) {
        for (int i = 0; i < Data.numVars; i++) {
            printf("%lf\t", data[j * Data.numVars + i]);
        }
        printf("\n");
    }
}

// Function to read CSV file and initialize data array
int readCSV(const char *filename) {
    FILE *file = fopen(filename, "r");

    if (file == NULL) {
        return -1;
    }

    // Read the first row to get variable names
    char line[1024];
    if (fgets(line, sizeof(line), file) != NULL) {
        // Tokenize the line to get variable names
        char *token = strtok(line, ",");
        int variableIndex = 0;

        while (token != NULL && variableIndex < MAX_VARIABLES) {
            // Remove leading and trailing whitespaces
            sscanf(token, " %[^ \t\n]", Data.variableNames[variableIndex]);

            // Print variable names if needed
            // printf("Variable %d: %s\n", variableIndex + 1, Data.variableNames[variableIndex]);

            token = strtok(NULL, ",");
            variableIndex++;
        }
        Data.numVars = variableIndex;
    }

    // Read the rest of the file to get numerical data
    int dataPointIndex = 0;
    while (fgets(line, sizeof(line), file) != NULL && dataPointIndex < MAX_DATA_POINTS) {
        char *token = strtok(line, ",");
        int variableIndex = 0;

        while (token != NULL && variableIndex < MAX_VARIABLES) {
            sscanf(token, " %d", &Data.data[dataPointIndex * Data.numVars + variableIndex]);

            // Print data if needed
            // printf("%s: %d\n", Data.variableNames[variableIndex], data[dataPointIndex * Data.numVars + variableIndex]);

            token = strtok(NULL, ",");
            variableIndex++;
        }

        dataPointIndex++;
    }
    Data.numObservations = dataPointIndex;

    fclose(file);
    return 0;
}


int copyDataToGPU(){
    int *gpu_data, *gpu_data_cpy;
    
    hipMalloc((void**)&gpu_data,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int));
    hipMalloc((void**)&gpu_data_cpy,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int));

    hipMemcpy(gpu_data,Data.data,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int),hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = ((MAX_VARIABLES*MAX_DATA_POINTS)+ blockSize - 1) / blockSize;

    copyArrayKernel<<<gridSize,blockSize>>>(gpu_data,gpu_data_cpy,(MAX_VARIABLES*MAX_DATA_POINTS));
    
    hipDeviceSynchronize();

    hipMemcpy(Data.data_cpy,gpu_data_cpy,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int),hipMemcpyDeviceToHost);
    
    // printCSVData(data_cpy);
    hipFree(gpu_data);
    hipFree(gpu_data_cpy);
    return 1;
}

int calculateVarSquared(int xvar){
    int *gpu_data;
    int *gpu_result;
    int sum=0;

    hipMalloc((void**)&gpu_data,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int));
    hipMemcpy(gpu_data,Data.data_cpy,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int),hipMemcpyHostToDevice);

    hipMalloc((void**)&gpu_result,sizeof(int));
    hipMemcpy(gpu_result,&sum,sizeof(int),hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = ((MAX_VARIABLES*MAX_DATA_POINTS)+ blockSize - 1) / blockSize;

    calculateSquareKernel<<<gridSize,blockSize>>>(gpu_data,gpu_result,(MAX_VARIABLES*MAX_DATA_POINTS),xvar);

    hipDeviceSynchronize();
    hipMemcpy(&sum,gpu_result,sizeof(int),hipMemcpyDeviceToHost);

    hipFree(gpu_data);
    hipFree(gpu_result);

    if(xvar==1)  globalCalculationInfo.sumSquaredX1=sum;
    if(xvar==2) globalCalculationInfo.sumSquaredX2=sum;

    return 0;
}


int calculateSumOfProductSquared(int var1, int var2){
    int *gpuData,*gpuResult;
    int sum=0;

    hipMalloc((void**)&gpuData,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int));
    hipMemcpy(gpuData,Data.data_cpy,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int),hipMemcpyHostToDevice);

    hipMalloc((void**)&gpuResult,sizeof(int));
    hipMemcpy(gpuResult,&sum,sizeof(int),hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = ((MAX_VARIABLES*MAX_DATA_POINTS)+ blockSize - 1) / blockSize;

    calculateSumOfProductSquaredKernel<<<gridSize,blockSize>>>(gpuData,gpuResult,(MAX_VARIABLES*MAX_DATA_POINTS),var1,var2);
    
    hipDeviceSynchronize();

    hipMemcpy(&sum,gpuResult,sizeof(int),hipMemcpyDeviceToHost);

    hipFree(gpuResult);
    hipFree(gpuData);

    if ((var1==0)&&(var2==1)) globalCalculationInfo.sumX1Y=sum;
    if ((var1==0)&&(var2==2)) globalCalculationInfo.sumX2Y=sum;
    if ((var1==1)&&(var2==2)) globalCalculationInfo.sumX1X2=sum;

    return 0;
}

int calculateVarSum(int var){
    int *gpu_data;
    int *gpu_result;
    int sum=0;

    hipMalloc((void**)&gpu_data,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int));
    hipMemcpy(gpu_data,Data.data_cpy,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int),hipMemcpyHostToDevice);

    hipMalloc((void**)&gpu_result,sizeof(int));
    hipMemcpy(gpu_result,&sum,sizeof(int),hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = ((MAX_VARIABLES*MAX_DATA_POINTS)+ blockSize - 1) / blockSize;

    calculateVarSumKernel<<<gridSize,blockSize>>>(gpu_data,gpu_result,(MAX_VARIABLES*MAX_DATA_POINTS),var);

    hipDeviceSynchronize();
    hipMemcpy(&sum,gpu_result,sizeof(int),hipMemcpyDeviceToHost);

    hipFree(gpu_data);
    hipFree(gpu_result);

    if (var==0) globalCalculationInfo.sumY=sum;
    if (var==1) globalCalculationInfo.sumX1=sum;
    if (var==2) globalCalculationInfo.sumX2=sum;
    return 0;
}

int predictModel(){
    int *gpuData;
    double *gpuPredictions,*gpuResiduals;

    hipMalloc((void**)&gpuData,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int));
    hipMalloc((void**)&gpuPredictions,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(double));
    hipMalloc((void**)&gpuResiduals,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(double));

    hipMemcpy(gpuData,Data.data_cpy,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(int),hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = ((MAX_VARIABLES*MAX_DATA_POINTS)+ blockSize - 1) / blockSize;

    predictModelKernel<<<gridSize,blockSize>>>(gpuData,gpuPredictions,gpuResiduals,globalRegressionInfo.beta_0,globalRegressionInfo.beta_1,globalRegressionInfo.beta_2,MAX_VARIABLES,Data.numObservations);

    hipDeviceSynchronize();

    hipMemcpy(Data.predictions,gpuPredictions,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(globalCalculationInfo.residuals,gpuResiduals,(MAX_VARIABLES*MAX_DATA_POINTS)*sizeof(double),hipMemcpyDeviceToHost);
    return 0;
}

int calculateStandardErrors(){
    return 0;
}
int calculateBetas(){
    double x1=globalCalculationInfo.sumSquaredX1-(pow(globalCalculationInfo.sumX1,2)/Data.numObservations);
    double x2=globalCalculationInfo.sumSquaredX2-(pow(globalCalculationInfo.sumX2,2)/Data.numObservations);
    double x1y=globalCalculationInfo.sumX1Y-(globalCalculationInfo.sumX1*globalCalculationInfo.sumY)/Data.numObservations;
    double x2y=globalCalculationInfo.sumX2Y-(globalCalculationInfo.sumX2*globalCalculationInfo.sumY)/Data.numObservations;
    double x1x2=globalCalculationInfo.sumX1X2-(globalCalculationInfo.sumX1*globalCalculationInfo.sumX2)/Data.numObservations;

    globalRegressionInfo.beta_1=((x2*x1y)-(x1x2*x2y))/((x1*x2)-pow(x1x2,2));
    globalRegressionInfo.beta_2=((x1*x2y)-(x1x2*x1y))/((x1*x2)-pow(x1x2,2));

    globalRegressionInfo.beta_0=(globalCalculationInfo.sumY/Data.numObservations)-
                                (globalRegressionInfo.beta_1*(globalCalculationInfo.sumX1/Data.numObservations))-
                                (globalRegressionInfo.beta_2*(globalCalculationInfo.sumX2/Data.numObservations));

    return 0;  
}

int runRegression(){
    printf("Running Regression...\n");
    calculateVarSquared(1);
    calculateVarSquared(2);
    calculateSumOfProductSquared(0,1);
    calculateSumOfProductSquared(0,2);
    calculateSumOfProductSquared(1,2);
    calculateVarSum(0);
    calculateVarSum(1);
    calculateVarSum(2);
    calculateBetas();
    predictModel();
    return 1;
}

int executeCommand(char command[]) {
    // exit
    if (strcmp(command, "e") == 0) {
        printf("Exiting program\n");
        return -1;
    }

    // Check if the first part of the command is "load"
    if (strncmp(command, "load", 4) == 0) {
        char file_loc[100];
        // Extract the rest of the command string after "load"
        sscanf(command, "%*s %s", file_loc);

        // successful read=0
        if (readCSV(file_loc) == 0) {
            printf("Read CSV file from: %s \n", file_loc);
            data_primed = true;
            copyDataToGPU();
        } else {
            printf("No such CSV file exists AND/OR Error in reading CSV File\n");
        }
        return 1;
    }

    // view
    if (strcmp(command, "view") == 0) {
        if (data_primed) {
            printCSVData(Data.data);
            printf("Number of vars: %d\nNumber of observations: %d\n",Data.numVars,Data.numObservations);
        } else {
            printf("Data is not loaded yet\n");
        }
        return 1;
    }


    if (strcmp(command,"def")==0){
        data_primed=true;
        readCSV("csv.csv");
        copyDataToGPU();
        runRegression();
        return 1;
    }    
    // Unrecognized command
    return 0;
}

int main() {
    char command[MAX_COMMAND_LENGTH];

    while (1) {
        // Print a prompt
        printf("$$> ");

        // Read a command from the user
        if (fgets(command, MAX_COMMAND_LENGTH, stdin) == NULL) {
            perror("Error reading command");
            exit(EXIT_FAILURE);
        }

        // Remove the newline character from the end of the command
        size_t length = strlen(command);
        if (length > 0 && command[length - 1] == '\n') {
            command[length - 1] = '\0';
        }

        // Execute the command using the system function
        int result = executeCommand(command);

        // Check if the command execution was successful
        if (result == 0) {
            printf("Unrecognized command\n");
        }

        // exit
        if (result == -1) {
            return 0;
        }
    }
    return 0;
}

